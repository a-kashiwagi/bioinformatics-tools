#include "hip/hip_runtime.h"
/******************************************************************************/
/*                                                                            */
/* Title  : Find to motif                                                     */
/* Program Name : fmotif.c                                                    */
/*                                                                            */
/* Detail : Make new                                                          */
/* Date   : 2010/06/06                                                        */
/* Author : Akihiro Kashiwagi                                                 */
/* e-mail : a-kashiwagi@mippi-mail.com                                        */
/*                                                                            */
/* Replace -------------------------------------------------------------------*/
/*                                                                            */
/* Date   : 2012/07/27                                                        */
/* Author : Akihiro Kashiwagi                                                 */
/* Deteil : Modifiy to threading                                              */
/*                                                                            */
/* Date   : 2014/01/05                                                        */
/* Author : Akihiro Kashiwagi                                                 */
/* Deteil : Modifiy for GPGUP                                                 */
/*                                                                            */
/* Date   :                                                                   */
/* Author :                                                                   */
/* Deteil :                                                                   */
/*                                                                            */
/*-------+---------+---------+---------+---------+---------+---------+--------*/
/*3456789012345678901234567890123456789012345678901234567890123456789012345678*/
/******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <ctype.h>
#include <unistd.h>

#include "/home/a-kashiwagi/cuda_by_example/common/book.h"

#define LOOP 1
#define BUFFER_SIZE 1024
#define THREADS 256
#define BLOCKS 1024
						/* Functions                  */
long ChkFileSize( char *filename );
int GetSequence( char *filename, char *buff );

__global__ void threading_procedure(
	char *seq,
	long  seq_num,
	long  src_cnt,
	long  lmer,
	long  lmt,
	long *humming_score,
	long *humming_num,
	char *humming_str
){

	char *lmer_src;				/* Source of lmer string      */
	char *lmer_trg;				/* Target of lmer string      */
	long  lmer_num;				/* Number of lmer             */
	long humming_now;			/* Current humming range      */
	long humming_bigst;			/* Biggest humming range      */
	long humming_cnt;			/* Counter of                 */
	long trg_cnt;				/* Counter of target          */
	long cnt;				/* Counter                    */
	char *str_dest;				/* for local strcpy           */
	char *str_src;                          /* for local strcpy           */

						/* Index for GPUs             */
	src_cnt += threadIdx.x + (blockIdx.x * blockDim.x);
	if( src_cnt >= seq_num - lmer ){
						/* Check for end sequence     */
		return;
	}
						/* Reset values               */
	humming_bigst = 0;
	humming_score[src_cnt] = 0;
	humming_num[src_cnt] = 0;
						/* Get source sequence        */
	lmer_src = &seq[src_cnt];
						/* Set lmer                   */
	str_dest = &humming_str[lmer * src_cnt];
	str_src  = lmer_src;
						/* strcpy                     */
	for(cnt = 0; cnt < lmer; cnt++){
		*str_dest++ = *str_src++;
	}
	//*str_dest = '\0';

	for( trg_cnt = 0; trg_cnt < seq_num - lmer; trg_cnt++ ){
						/* Loop for target sequence   */

		if( src_cnt == trg_cnt ){
						/* Continue to                */
						/*          equality sequence */
			continue;
		}
						/* Get target lmer            */
		lmer_trg = &seq[trg_cnt];

						/* Reset humming number       */
		humming_now = 0;
						/* Calculate to humming range */
		for(	humming_cnt = 0;
			humming_cnt < lmer;
			humming_cnt++
		){
						/* Equality Charactor         */
						/*          source and target */
			if( lmer_src[humming_cnt]
				== lmer_trg[humming_cnt] ){

				humming_now++;
			}			/* Increment counter          */
		}
						/* Find to                    */
						/*      bigger score sequence */
		if( humming_now > humming_bigst ){
						/* Change from last to now    */
			humming_bigst = humming_now;
						/* Store to now number        */
			humming_score[src_cnt] = humming_now;
			humming_num[src_cnt] = 1;

		}else if( humming_now == humming_bigst ){

			humming_num[src_cnt]++;
						/* Find to equality score,    */
						/*     then increment counter */
		}
	}

						/* Check for limit            */
	/*
	if( humming_score[src_cnt] >= lmt ){

		printf(	"%ld,%ld,%s,%ld,%ld\n",
			src_cnt + 1,
			src_cnt + args->lmer,
			args->hs[src_cnt].str,
			args->hs[src_cnt].humming_score,
			args->hs[src_cnt].humming_num
		);
	}
	*/

	return;
}
						/* Function of main()         */
int main(int argc, char *argv[]){

	int ret;				/* Number of return           */
	long cnt;				/* Counter                    */
	long lmer;				/* Number of lmer             */
	long lmt;				/* Limit to score             */
	long ctn;				/* Continue from this         */
	char filename[BUFFER_SIZE];		/* Sequence fine name         */
	long filesize;				/* Size of file               */
	long seq_num;				/* Number of sequence         */
	long src_cnt;				/* Counter of source          */
						/*      calculating humming   */

	long *humming_score;			/* Score of humming range     */
	long *humming_num;			/* Number of equality score   */
	char *humming_str;			/* String of lmer             */
	char *endptr;
	char *seq;

						/* Pointer for GPGPU          */
	char *seq_dev;
						/* For Sequence               */
	long *hscr_dev;
						/* For Humming score          */
	long *hnum_dev;
						/* For Humming number         */
	char *hstr_dev;
						/* For Humming string         */
	lmer = 9;
	lmt  = 0;
	ctn  = 0;
						/* Check for argument         */
	if( argc > 1 ){
		for( cnt = 1; cnt < argc; cnt++ ){

						/* Get lmer                   */
			if( strcmp( argv[cnt], "-lmer" ) == 0 ){

				lmer = strtol( argv[cnt + 1], &endptr, 0);
				if( *endptr != '\0' ){
						/* Error                      */
					printf("Invalid number for -lmer.\n");
					exit(-1);
				}

						/* Get limit score            */
			}else if( strcmp( argv[cnt], "-lmt" ) == 0 ){

				lmt = strtol( argv[cnt + 1], &endptr, 0);
				if( *endptr != '\0' ){
						/* Error                      */
					printf("Invalid number for -lmt.\n");
					exit(-1);
				}

						/* Get continue location num  */
			}else if( strcmp( argv[cnt], "-ctn" ) == 0 ){

				ctn = strtol( argv[cnt + 1], &endptr, 0);
				if( *endptr != '\0' ){
						/* Error                      */
					printf("Invalid number for -ctn.\n");
					exit(-1);
				}
				ctn--;
			}
		}
						/* Get fine name              */
		strcpy( filename, argv[argc - 1] );

	}else{
						/* Not enough arguments       */
		printf("  Usage: fmotif [ Options ] seqence_filename\n");
		printf("Options:\n");
		printf("  -lmer number : set lmer - default 9\n");
		printf("  -lmt  number : set lower limit of lmer");
		printf(" - default 0\n");
		printf("  -ctn  number : set continue from this - default 0\n");
		printf("\n");

		exit(0);
	}
						/* Get file size              */
	filesize = ChkFileSize( filename );
	if( filesize != 0 ){
						/* Malloc for sequence        */
		seq = (char *)malloc( sizeof(int) * filesize );
		if( seq == NULL ){
			printf("Can not allocate memory.\n");
			exit(-1);
		}
						/* Set sequence number        */
		seq_num = filesize;
						/* Malloc for                 */
						/*       structure of humming */
		humming_score = (long *)malloc( sizeof(long) * seq_num);
		humming_num   = (long *)malloc( sizeof(long) * seq_num);
		humming_str   = (char *)malloc( sizeof(char) * lmer * seq_num);

	}else{
						/* Case of void               */
		printf("File is void.\n");
		exit(1);
	}
						/* Get sequence               */
	ret = GetSequence( filename, seq );

						/* Find to motif              */

						/* Set count of start location*/
	src_cnt = ctn;
						/* Memory allocation at GPU   */

	HANDLE_ERROR(hipMalloc( (void**)&seq_dev,  sizeof(int)  * filesize ));
						/* For sequence               */

	HANDLE_ERROR(hipMalloc( (void**)&hscr_dev, sizeof(long) * seq_num  ));
						/* For humming score          */

	HANDLE_ERROR(hipMalloc( (void**)&hnum_dev, sizeof(long) * seq_num  ));
						/* For humming numger         */

	HANDLE_ERROR(
		hipMalloc( (void**)&hstr_dev, sizeof(long) * lmer * seq_num  )
	);
						/* For humming string         */

						/* Data copy to GPGPU         */
	HANDLE_ERROR(
		hipMemcpy(
			seq_dev,
			seq,
			sizeof(int) * filesize,
			hipMemcpyHostToDevice
		)
						/* Sequence                   */
	);

	HANDLE_ERROR(
		hipMemcpy(
			hscr_dev,
			humming_score,
			sizeof(long) * seq_num,
			hipMemcpyHostToDevice
		)
						/* Humming score              */
	);

	HANDLE_ERROR(
		hipMemcpy(
			hnum_dev,
			humming_num,
			sizeof(long) * seq_num,
			hipMemcpyHostToDevice
		)
						/* Humming number             */
	);

	HANDLE_ERROR(
		hipMemcpy(
			hstr_dev,
			humming_str,
			sizeof(char) * lmer * seq_num,
			hipMemcpyHostToDevice
		)
						/* Humming string             */
	);

	while(LOOP){
						/* Call function of GPU side  */
		threading_procedure<<<BLOCKS,THREADS>>>(
			seq_dev,
			seq_num,
			src_cnt,
			lmer,
			lmt,
			hscr_dev,
			hnum_dev,
			hstr_dev
		);

		if(src_cnt >= seq_num - lmer ){
						/* Check end sequence         */
			break;
		}
						/* Set next start location    */
		src_cnt += BLOCKS * THREADS;
	}

						/* Data copy from GPGPU       */
	HANDLE_ERROR(
		hipMemcpy(
			humming_score,
			hscr_dev,
			sizeof(long) * seq_num,
			hipMemcpyDeviceToHost
		)
						/* Sequence                   */
	);

	HANDLE_ERROR(
		hipMemcpy(
			humming_num,
			hnum_dev,
			sizeof(long) * seq_num,
			hipMemcpyDeviceToHost
		)
						/* Humming number             */
	);

	HANDLE_ERROR(
		hipMemcpy(
			humming_str,
			hstr_dev,
			sizeof(char) * lmer * seq_num,
			hipMemcpyDeviceToHost
		)
						/* Humming string             */
	);
						/* Free to GPU memory         */
	HANDLE_ERROR(hipFree(  seq_dev ));
	HANDLE_ERROR(hipFree( hscr_dev ));
	HANDLE_ERROR(hipFree( hnum_dev ));
	HANDLE_ERROR(hipFree( hstr_dev ));
						/* Output                     */
	long cnt_lmer;

	for(cnt = 0; cnt < seq_num - lmer; cnt++){

		if( !(humming_score[cnt] >= lmt) ){
						/* Check for limit            */
			continue;
		}

		printf("%d,%d,",cnt + 1,cnt + lmer);

		for(cnt_lmer = 0; cnt_lmer < lmer; cnt_lmer++){
			printf("%c",humming_str[(lmer * cnt) + cnt_lmer]);
		}

		printf(",%ld,%ld\n", humming_score[cnt], humming_num[cnt]);
	}
						/* Free to memory             */
	free( humming_score );
	free( humming_num   );
	free( humming_str   );
	free( seq );
						/* Normal terminate           */
	exit(0);
}

/******************************************************************************/
/*                                                                            */
/* Title  : Check for filesize function                                       */
/* Function Name : ChkFileSize                                                */
/*                                                                            */
/* Detail :                                                                   */
/* Date   : 2009/12/03                                                        */
/* Author : Akihiro Kashiwagi                                                 */
/* E-mail : a-kashiwagi@mippi-mail.com                                        */
/*                                                                            */
/* Input  : char *filename : Target file name                                 */
/*                                                                            */
/* Output : int : [ Byte : Normal Terminate ] [ -1 : Error ]                  */
/*                                                                            */
/* Replace -------------------------------------------------------------------*/
/*                                                                            */
/* Date   :                                                                   */
/* Author :                                                                   */
/* Deteil :                                                                   */
/*                                                                            */
/*-------+---------+---------+---------+---------+---------+---------+--------*/
/*3456789012345678901234567890123456789012345678901234567890123456789012345678*/
/******************************************************************************/

long ChkFileSize( char *filename ){

        struct stat st;                         /* Struct Stat                */
        
        stat( filename, &st );                  /* Get status                 */
        
        return( st.st_size );                   /* Return file size           */
}

/******************************************************************************/
/*                                                                            */
/* Title  : Get sequence function                                             */
/* Function Name : GetSequence                                                */
/*                                                                            */
/* Detail :                                                                   */
/* Date   : 2009/12/03                                                        */
/* Author : Akihiro Kashiwagi                                                 */
/* E-mail : a-kashiwagi@mippi-mail.com                                        */
/*                                                                            */
/* Input  : char *filename : Target file name                                 */
/*          char *buff : Buffer of data of sequence                           */
/*                                                                            */
/* Output : int : [ 0 : Normal Terminate ] [ -1 : Error ]                     */
/*                                                                            */
/* Replace -------------------------------------------------------------------*/
/*                                                                            */
/* Date   : 2010/08/24                                                        */
/* Author : Akihiro Kashiwagi                                                 */
/* Deteil : Added procedure that convert to upper character                   */
/*                                                                            */
/* Date   :                                                                   */
/* Author :                                                                   */
/* Deteil :                                                                   */
/*                                                                            */
/*-------+---------+---------+---------+---------+---------+---------+--------*/
/*3456789012345678901234567890123456789012345678901234567890123456789012345678*/
/******************************************************************************/

int GetSequence( char *filename, char *buff ){

        FILE *fp;                               /* File pointer               */
        long cnt;                               /* Counter                    */
        long in_char;                           /* In character               */
        
        fp = fopen( filename, "r" );            /* File open                  */
        if( fp == NULL ){                       /* Error                      */
                return(-1);
        }
        
        /* buff[0] = ' '; */                    /* Insert space at header     */
        
        in_char = fgetc( fp );                  /* Get a character from file  */
        cnt = 0;                                /* Counter initialize         */
        
        while( in_char != EOF ){
        
		in_char = toupper( in_char );	/* Convert to upper           */

                if( in_char != '\n' ){          /* LF skip                    */
                        buff[cnt] = in_char;    /* Copy to buffer             */
                        cnt++;                  /* Counter increment          */
                }
                
                in_char = fgetc( fp );          /* Get next character         */
        }
        
        buff[cnt] = '\0';                       /* Set null at footer         */
        
        fclose( fp );                           /* File close                 */
        return(0);                              /* Normal terminate           */
}


